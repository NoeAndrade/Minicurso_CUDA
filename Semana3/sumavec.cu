#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void vectorSuma(double *A, double *B, double *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i];
  }
}


int main()
{
    hipError_t err = hipSuccess;
    double *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
    int N=100;
    h_A=new double[N];
    h_B=new double[N];
    h_C=new double[N];
    int size=sizeof(double)*N;

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i<N; ++i) {
        h_A[i] = rand() / (double)RAND_MAX;
        h_B[i] = rand() / (double)RAND_MAX;
        
    }

    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy vector A from host to device (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy vector B from host to device (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
       
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorSuma<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    err = hipGetLastError();
  
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy vector B from host to device (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < N; ++i) {
        fprintf(stderr, "h_A[%d]: %f, h_B[%d]: %f, h_C[%d]: %f\n", i,h_A[i], i,h_B[i], i,h_C[i]);
      }
return 0;

}
