#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "cuda_Error.cuh"
__global__ void vectorSuma(double *A, double *B, double *C, int numElements) //Definiendo el Kernel
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= numElements)return; //Los hilos que no son menores que numElements no realizan los siguientes calculos

  C[i] = A[i] + B[i];
}


int main()
{
    hipError_t err = hipSuccess;
    double *h_A,*h_B,*h_C,*d_A,*d_B,*d_C; //arreglos

    int N=100; //numero de elementos que contiene el arreglo

    h_A = new double[N];
    h_B = new double[N];
    h_C = new double[N];

    size_t size=sizeof(double)*N; //numero de bytes que requiero para los arreglos

    //comprobando que la asignación de memoria de los arreglos en el Host fue exitosa
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "No se pudo asignar memoria para los vectores en el host !\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i<N; ++i) {
        h_A[i] = rand() / (double)RAND_MAX;
        h_B[i] = rand() / (double)RAND_MAX;
        
    }
    //El segundo parametro del Errorcuda es 0 ya que este corresponde a reviasr si se asigna memoria con hipMalloc correctamente.
    Errorcuda(hipMalloc(&d_A, size),0,"d_A"); 
    Errorcuda(hipMalloc(&d_B, size),0,"d_B"); 
    Errorcuda(hipMalloc(&d_C, size),0,"d_C"); 

    //Se copia la información que se asigno en el Host a h_A y h_B y se copia en la memoria global de la GPU en d_A y d_B respectivamente
    Errorcuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice),1,"h_A");
    Errorcuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice),1,"h_B");
       
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorSuma<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    err = hipGetLastError();
  
    Errorcuda(err,3,"vectorSuma");
    
    Errorcuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost),2,"d_C");
    //Las operaciones de hipMemcpy sincronizan el device por lo que no es necesario usar cudadeviceSynchronize()
    
    Errorcuda(hipFree(d_A),4,"d_A");
    Errorcuda(hipFree(d_B),4,"d_B");
    Errorcuda(hipFree(d_C),4,"d_C");
    
    for (int i = 0; i < N; ++i) {
      fprintf(stderr, "h_A[%d]: %f, h_B[%d]: %f, h_C[%d]: %f\n", i,h_A[i], i,h_B[i], i,h_C[i]);
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;  

    return 0;

}
