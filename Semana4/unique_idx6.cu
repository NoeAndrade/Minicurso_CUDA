//cuda libraries//
#include "hip/hip_runtime.h"

#include <stdio.h>
//kernel para la suma de vectores

__global__ void unique_gid_calculation_2d(int * input)
{
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	int gid = tid + block_offset + row_offset;
	printf("blockIdx : %d, threadIdx : %d ,gid : %d, value : %d \n",blockIdx.x, tid,gid,input[gid]);

}



// main suma de vectores
int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,9,4,53,65,12,1,33,11,12,13,14,15,16,17,18};

	for (int i=0; i < array_size; i++)
	{
		printf("%d ",h_data[i]);			
	}
	printf("\n \n");
	
	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data,h_data, array_byte_size , hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2,2);

	unique_gid_calculation_2d <<< grid, block >>> (d_data);

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


