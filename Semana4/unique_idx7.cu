//cuda libraries//
#include "hip/hip_runtime.h"

#include <stdio.h>
//kernel para la suma de vectores

__global__ void unique_gid_calculation_2d(int * input)
{
	int tid = blockDim.x * threadIdx.y + threadIdx.x;
	
	int num_threads_in_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_threads_in_block;

	int num_threads_in_row = num_threads_in_block * gridDim.x;
	int row_offset = num_threads_in_row* blockIdx.y;

	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx : %d ,gid : %d, value : %d \n",blockIdx.x,blockIdx.y, tid,gid,input[gid]);

}



// main suma de vectores
int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,9,4,53,65,12,1,33,11,12,13,14,15,16,17,18};

	for (int i=0; i < array_size; i++)
	{
		printf("%d ",h_data[i]);			
	}
	printf("\n \n");
	
	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data,h_data, array_byte_size , hipMemcpyHostToDevice);

	dim3 block(2,2);
	dim3 grid(2,2);

	unique_gid_calculation_2d <<< grid, block >>> (d_data);

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


