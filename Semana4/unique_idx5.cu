//cuda libraries//
#include "hip/hip_runtime.h"

#include <stdio.h>
//kernel para la suma de vectores
__global__ void unique_idx_calc_threadIdx(int * input)
{
	int tid = threadIdx.x;
	printf("threadIdx : %d, value : %d \n" ,tid, input[tid]);

}

__global__ void unique_gid_calculation(int * input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx : %d, threadIdx : %d ,gid : %d, value : %d \n",blockIdx.x, tid,gid,input[gid]);

}



// main suma de vectores
int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,9,4,53,65,12,1,33,11,12,13,14,15,16,17,18};

	for (int i=0; i < array_size; i++)
	{
		printf("%d ",h_data[i]);			
	}
	printf("\n \n");
	
	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data,h_data, array_byte_size , hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2,2);

	unique_gid_calculation <<< grid, block >>> (d_data);

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


