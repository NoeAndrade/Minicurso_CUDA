#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//for memset
#include <cstring>


__global__ void sum_array(int * a, int * b, int * c, int size)
{
	int gid = blockIdx.x * blockDim.x +threadIdx.x;

	if (gid < size)
	{
		c[gid] =a[gid] + b[gid];
	}

}

int main(){
	int size = 10000;
	int block_size = 128;

	int NO_BYTES = size * sizeof(int);

	//host pointers
	int* h_a, *h_b, *gpu_results;

	h_a = (int*)malloc(NO_BYTES);
	h_b = (int*)malloc(NO_BYTES);
	gpu_results = (int*)malloc(NO_BYTES);


	//Inicializamos host pointer
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0;i<size;i++)
	{
		h_a[i] = (int)(rand() & 0xFF);
	}
	for (int i = 0;i<size;i++)
        {               
                h_b[i] = (int)(rand() & 0xFF);
        }

	memset(gpu_results,0,NO_BYTES);

	//Device pointer
	int* d_a, *d_b, *d_c;
	hipMalloc((int **)&d_a, NO_BYTES);
	hipMalloc((int **)&d_b, NO_BYTES);
	hipMalloc((int **)&d_c, NO_BYTES);
	//Transferencia de memoria from host to device
	hipMemcpy(d_a, h_a,NO_BYTES,hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,NO_BYTES,hipMemcpyHostToDevice);
	
	//lanzamos el grid
	dim3 block(block_size);
	dim3 grid((size/block.x)+1);
	
	sum_array<<<grid,block>>> (d_a,d_b,d_c,size);
	hipDeviceSynchronize();


	hipMemcpy(gpu_results, d_c,NO_BYTES,hipMemcpyDeviceToHost);
	for (int i=0;i<size;i++)
	{

	printf("C[%d] : %d \n",i,gpu_results[i]);
	}
}




