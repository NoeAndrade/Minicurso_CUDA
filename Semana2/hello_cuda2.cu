//cuda libraries//
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello world in cuda \n");
}


int main()
{
	dim3 block(4);
    	dim3 grid(8);
    	hello_cuda<<<grid, block>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


