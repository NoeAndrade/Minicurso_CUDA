//cuda libraries//
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello world in cuda \n");
}


int main()
{
	hello_cuda<<<1,1>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


